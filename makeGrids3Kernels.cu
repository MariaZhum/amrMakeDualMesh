#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2021 Ingo Wald, 2023 Maria Zhumabaeva                     //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "umesh/UMesh.h"
#include "umesh/io/IO.h"
#include "umesh/check.h"
// #include "tetty/UMesh.h"
#include <cstring>
#include <set>
#include <map>
#include <fstream>
#include <atomic>
#include <array>
#include <chrono>
#include "timer.h"
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <iomanip>


#ifndef PRINT
#define PRINT(var) std::cout << #var << "=" << var << std::endl;
#ifdef __WIN32__
#define PING std::cout << __FILE__ << "::" << __LINE__ << ": " << __FUNCTION__ << std::endl;
#else
#define PING std::cout << __FILE__ << "::" << __LINE__ << ": " << __PRETTY_FUNCTION__ << std::endl;
#endif
#endif


using namespace umesh;
using namespace std::chrono;

const int macroCellWidth = 8;
const bool PRINT_EVERY_BRICK_SCALAR = false;
const bool PRINT_STAT = false;

template <typename T>
inline T __host__ __device__ iDivUp(T a, T b){
  return (a+b-1) / b;
}

struct Cube{
  vec3f lower;
  int level;
  std::array<int, 8> scalarIDs;
};

struct Brick{
  Brick(int lvl){
    level = lvl;
  }

  box3i dbg_bounds;

  __device__ void setAttributes(box3i &bounds, int offsetFirst, int offsetLast){
    dbg_bounds = bounds;
    lower = bounds.lower;

    numCubes.x = bounds.upper.x - bounds.lower.x;
    numCubes.y = bounds.upper.y - bounds.lower.y;
    numCubes.z = bounds.upper.z - bounds.lower.z;

    offset = offsetFirst;
    numScalars = offsetLast - offsetFirst;
  }

  vec3i lower;
  int level;
  vec3i numCubes;
  int *scalarIDs;
  int offset;
  int numScalars;
};

vec3i make_vec3i(vec3f v) { return {int(v.x), int(v.y), int(v.z)}; }
vec3f make_vec3f(vec3i v) { return {float(v.x), float(v.y), float(v.z)}; }

vec3i cellID(const Cube &cube){
  vec3i cid = make_vec3i(cube.lower);
  if (cube.lower.x < 0.f)
    cid.x -= ((1 << cube.level) - 1);
  if (cube.lower.y < 0.f)
    cid.y -= ((1 << cube.level) - 1);
  if (cube.lower.z < 0.f)
    cid.z -= ((1 << cube.level) - 1);
  cid = cid / (1 << cube.level);

  // if (cid == vec3i(-1,-1,-1)) {
  //   PING;
  //   PRINT(cube.lower);
  //   PRINT(cid);
  // }
  return cid;
}

box3i cellBounds(const Cube &cube){
  vec3i cell = cellID(cube);
  return {cell, cell + vec3i(1)};
}

vec3i mcID(const Cube &cube){
  vec3i cid = cellID(cube);
  if (cid.x < 0)
    cid.x -= (macroCellWidth - 1);
  if (cid.y < 0)
    cid.y -= (macroCellWidth - 1);
  if (cid.z < 0)
    cid.z -= (macroCellWidth - 1);
  vec3i mcid = cid / macroCellWidth;
  // if (mcid == vec3i(-1,-1,-1)) {
  //   PING;
  //   PRINT(cube.lower);
  //   PRINT(cellID(cube));
  //   PRINT(mcid);
  // }
  return mcid;
}

__device__ void calcMCID(int &mcIDx, int &mcIDy, int &mcIDz, int cellIDx, int cellIDy, int cellIDz){
  mcIDx = cellIDx;
  mcIDy = cellIDy;
  mcIDz = cellIDz;

  if (cellIDx < 0)
    mcIDx -= (macroCellWidth - 1);
  if (cellIDy < 0)
    mcIDy -= (macroCellWidth - 1);
  if (cellIDz < 0)
    mcIDz -= (macroCellWidth - 1);

  mcIDx = mcIDx / macroCellWidth;
  mcIDy = mcIDy / macroCellWidth;
  mcIDz = mcIDz / macroCellWidth;
}

__device__ void calcCellID(int &cellIDx, int &cellIDy, int &cellIDz, vec3f lower, int level){
  cellIDx = (int)lower.x;
  cellIDy = (int)lower.y;
  cellIDz = (int)lower.z;

  if (lower.x < 0.f)
    cellIDx -= ((1 << level) - 1);
  if (lower.y < 0.f)
    cellIDy -= ((1 << level) - 1);
  if (lower.z < 0.f)
    cellIDz -= ((1 << level) - 1);

  cellIDx = cellIDx / (1 << level);
  cellIDy = cellIDy / (1 << level);
  cellIDz = cellIDz / (1 << level);
}

box3f worldBounds(const Brick &brick){
  box3f bb;
  bb.lower = make_vec3f(brick.lower * (1 << brick.level));
  bb.upper = bb.lower + make_vec3f(brick.numCubes * (1 << brick.level));
  return bb;
}

vec3i getLevelSizeInMC(vec3f &levelLower, vec3f &levelUpper, int level){
  Cube minCube;
  Cube maxCube;

  minCube.lower = levelLower;
  minCube.level = level;

  maxCube.lower = levelUpper;
  maxCube.level = level;

  return mcID(maxCube) - mcID(minCube) + vec3i(1);
}

// kernel 1
/* calculates bounds for each macrocell depending on cubes(given by their lower coord.)
   and number of cubes in each macrocell
*/
__global__ void setBoundsAndCubes(vec3f *cubesLower, vec3i levelSizeInMC, vec3i levelLower, int *listOfcubesIDXsforMC,
                                  box3i *mcBounds, int level, int *offsetsCubes, int totalNumOfCubes){

  int cubeNum = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (cubeNum < totalNumOfCubes)
  {
    //printf("cubeNum = %d (kernel1 - inside if ) \n", cubeNum);
    int cellIDx, cellIDy, cellIDz;
    calcCellID(cellIDx, cellIDy, cellIDz, cubesLower[cubeNum], level);

    int mcIDx, mcIDy, mcIDz;
    calcMCID(mcIDx, mcIDy, mcIDz, cellIDx, cellIDy, cellIDz);

    int linearMcIDX = mcIDx-levelLower.x  + (mcIDy-levelLower.y) * levelSizeInMC.x + (mcIDz-levelLower.z) * levelSizeInMC.x * levelSizeInMC.y;
  
    // extend bounds
    // min function is associative
    atomicMin(&mcBounds[linearMcIDX].lower.x, cellIDx);
    atomicMin(&mcBounds[linearMcIDX].lower.y, cellIDy);
    atomicMin(&mcBounds[linearMcIDX].lower.z, cellIDz);

    atomicMax(&mcBounds[linearMcIDX].upper.x, cellIDx + 1);
    atomicMax(&mcBounds[linearMcIDX].upper.y, cellIDy + 1);
    atomicMax(&mcBounds[linearMcIDX].upper.z, cellIDz + 1);
    
    int prevOffsetCubes = atomicAdd(&offsetsCubes[linearMcIDX], 1);
 
    listOfcubesIDXsforMC[linearMcIDX * (macroCellWidth * macroCellWidth * macroCellWidth) + prevOffsetCubes] = cubeNum;
    
  }
}

// kernel 2
/*  calculates the max number of scalars for each macrocell
*/
__global__ void calcMaxNumOfScalars(int numOfMC, box3i *mcBounds, int *maxNumOfScalars){
  int brickNum = blockIdx.x * blockDim.x + threadIdx.x;
  //printf("bricknum = %d", brickNum);
  if (brickNum < numOfMC){
    int x = mcBounds[brickNum].upper.x - mcBounds[brickNum].lower.x;
    int y = mcBounds[brickNum].upper.y - mcBounds[brickNum].lower.y;
    int z = mcBounds[brickNum].upper.z - mcBounds[brickNum].lower.z;
    maxNumOfScalars[brickNum] = (x+1) * (y+1) * (z+1);
  }
}

/*  writes scalars of each cube into one array 
    index within array is determined by linear brick number and the number 
    of scalars in previous bricks
*/
__device__ void writeCube(vec3f &cubeLower, Brick &brick, int *scalars, int offset, int *resultScalarArray){
  int cellIDx, cellIDy, cellIDz;

  calcCellID(cellIDx, cellIDy, cellIDz, cubeLower, brick.level);
  // printf("cellID (%d, %d, %d)\n", cellIDx, cellIDy, cellIDz);
  int baseX = cellIDx - brick.lower.x;
  int baseY = cellIDy - brick.lower.y;
  int baseZ = cellIDz - brick.lower.z;

  int vtkOrder[8] = {0, 1, 3, 2, 4, 5, 7, 6};

  // index within brickx, worldSizeInMC.y, worldSizeInMC.z
  int idx;

  for (int iz = 0; iz < 2; iz++)
    for (int iy = 0; iy < 2; iy++)
      for (int ix = 0; ix < 2; ix++){
        idx = baseX + ix + (brick.numCubes.x + 1) * (baseY + iy + (brick.numCubes.y + 1) * (baseZ + iz));
        resultScalarArray[idx + offset] = scalars[vtkOrder[4 * iz + 2 * iy + ix]];
      }
}

/* sets scalars for a given brick (=start position in resultScalarsArr)
   to -1 == empty cell
*/
__device__ void setResultScalarArrToEmpty(int *resultScalarsArr, int start, int end){
  for (int i = start; i <= end; i++){
    resultScalarsArr[i] = -1;
  }
}

// kernel 3
/*  creates brick by writing its attributes 
    (brick is empty if brick.numCubes == 0)
    and writes scalars of that brick into resultScalarsArr
*/
__global__ void createAndFillBricks(vec3f *cubesLower, box3i *mcBounds, int *listOfcubesIDXsforMC, int *scalars, Brick *mcBricks,
                                int level, int *offsetCubes, int totalNumOfMC, int *offsetScalars, int *resultScalarsArr){
  
  int brickNum = blockIdx.x * blockDim.x + threadIdx.x;

  if (brickNum < totalNumOfMC && offsetCubes[brickNum]!=0){
    //set all scalars to empty = -1
    setResultScalarArrToEmpty(resultScalarsArr, offsetScalars[brickNum], offsetScalars[brickNum+1] - 1);

    mcBricks[brickNum].setAttributes(mcBounds[brickNum], offsetScalars[brickNum], offsetScalars[brickNum+1]);

    int scalarsCube[8];
    int cubeidx;

     // for each cube in MC write its scalars into resultScalarsArr
    for (int i = 0; i < offsetCubes[brickNum]; i++){
      cubeidx = listOfcubesIDXsforMC[brickNum * macroCellWidth * macroCellWidth * macroCellWidth + i];

      #pragma unroll
      for (int j = 0; j < 8; j++){
        scalarsCube[j] = scalars[cubeidx * 8 + j];
      }

      writeCube(cubesLower[cubeidx], mcBricks[brickNum], scalarsCube, offsetScalars[brickNum], resultScalarsArr);
    }    
  }
}

// writes statistics for one level
void writeStat(int level, int numOfBricks, int numOfCubes, float kernel1, float kernel2, float kernel3, float totalKernelTime, float thrustPrefSum, float step1, float step2, float step3, float totalStepTime){
  std::ofstream outFile;
  outFile.open ("stat.txt", std::ofstream::out | std::ofstream::app);

  if(!outFile.is_open()){
    std::cout << "Error opening file!" << std::endl;
    return;
  }

  outFile << "level = " << level << ", number of generated cubes = " << numOfCubes-1 << ", number of generated bricks = " << numOfBricks << std::endl;

  outFile << "+" << std::setfill('-') << std::setw(20) << "+" << std::setw(24) << "+" << std::setw(18) << "+" << std::endl;
  outFile << std::left << std::setfill(' ') << std::setw(20)<< "|" << "|" << std::setw(23) << "GPU (incl. alloc/cpy)" << "|" << std::setw(17) << "GPU (kernel only)" <<"|" << std::endl;
  outFile << std::setfill('-') << std::setw(20) << "+"  << std::setw(24) << "+"  << std::setw(18) << "+" << "+" << std::endl;

  outFile << std::left << "|" << std::setfill(' ') << std::setw(19) << "step/kernel 1" << "|" << std::setw(23) << step1 << "|" << std::setw(17) << kernel1 << "|"<< std::endl;
  outFile << std::left << "|" << std::setfill(' ') << std::setw(19) << "step/kernel 2" << "|" << std::setw(23) <<  step2 << "|" << std::setw(17) << kernel2 << "|"<< std::endl;
  outFile << std::left << "|" << std::setfill(' ') << std::setw(19) << "thrust" << std::setw(24) << "|" << "|" << std::setw(17) << thrustPrefSum<< "|"<< std::endl;
  outFile << std::left << "|" << std::setfill(' ') << std::setw(19) << "step/kernel 3"<< "|" << std::setw(23) << step3 << "|" << std::setw(17) << kernel3 << "|"<< std::endl;
  outFile << std::left << "|" << std::setfill(' ') << std::setw(19) << "total"<< "|" << std::setw(23) << totalStepTime<< "|" << std::setw(17) << totalKernelTime << "|"<< std::endl;

  outFile << std::setfill('-') << std::setw(20) << "+"  << std::setw(24) << "+"  << std::setw(18) << "+" << "+" << std::endl;

  outFile << " " << std::endl;
  outFile << " " << std::endl;

  outFile.close();

}

/*! the 'cells' are all in a space where each cell is exactly 1
    int-coord wide, so the second cell on level 1 is _not_ at
    (2,2,2)-(4,4,4), but at (1,1,1)-(2,2,2). To translate from this
    level-L cell space to world coordinates, take cell (i,j,k) and get
    lower=((i,j,k)+.5f)*(1<<L), and upper = lower+(1<<L) */
std::vector<Brick> makeBricksForLevel(int level,
                                      std::vector<Cube> &cubes, int *&resultScalarArray){
  auto start = high_resolution_clock::now();

  gridlets::timer t;

  size_t numOfCubes = cubes.size();

  // lower and upper .lower point of cubes for current lvl in world coord 
  vec3f levelLower = vec3f(FLT_MAX, FLT_MAX, FLT_MAX);
  vec3f levelUpper = vec3f(FLT_MIN, FLT_MIN, FLT_MIN);

  std::vector<vec3f> cubesLower;
  cubesLower.reserve(numOfCubes);

  std::vector<int> scalarsArray;
  scalarsArray.reserve(8*numOfCubes);

  t.reset();

  for (auto cube: cubes){
    levelLower = min(levelLower, cube.lower);
    levelUpper = max(levelUpper, cube.lower);
    cubesLower.push_back(cube.lower);
    for (int j = 0; j < 8; j++){
      scalarsArray.push_back(cube.scalarIDs[j]);
    }
  }

  std::cout << __LINE__ << " " << t.elapsed() << "s time for copying .lower and .scalars into arrays and finding min/max\n"
            << std::endl;
  t.reset();

 //for shifting MC Grid to the point of origin since we use [0] as the starting index and not the actual macrocell id 
  Cube lowestCube;
  lowestCube.lower = levelLower;
  lowestCube.level = level;

  // size of grid in mc determined by cubes
  vec3i levelSizeInMC = getLevelSizeInMC(levelLower, levelUpper, level);

  size_t numberOfMC = levelSizeInMC.x * levelSizeInMC.y * levelSizeInMC.z;
  
  t.reset();

  // 1st kernel
  std::vector<box3i> mcBounds;
  mcBounds.resize(numberOfMC);

  // alloc mem device
  box3i *ptr_mcBounds;
  int *ptr_listOfcubesIDXsforMC;
  vec3f *ptr_cubesLower;
  int *ptr_offsetsCubes;

  t.reset();
  hipDeviceSynchronize();
  std::cout << __LINE__ << " " << t.elapsed() << "s time for setting up cuda \n"
            << std::endl;
  t.reset();

  hipMalloc((void **)&ptr_mcBounds, numberOfMC * sizeof(box3i));
  hipMalloc((void **)&ptr_listOfcubesIDXsforMC, numberOfMC * (macroCellWidth * macroCellWidth * macroCellWidth) * sizeof(int));
  hipMalloc((void **)&ptr_cubesLower, numOfCubes * sizeof(vec3f));
  hipMalloc((void **)&ptr_offsetsCubes, numberOfMC * sizeof(int));
  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 1 alloc. \n"
            << std::endl;
  t.reset();

  hipMemcpy(ptr_mcBounds, &mcBounds[0], numberOfMC * sizeof(box3i), hipMemcpyHostToDevice);
  hipMemcpy(ptr_cubesLower, &cubesLower[0], numOfCubes * sizeof(vec3f), hipMemcpyHostToDevice);
  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 1 copy\n"
            << std::endl;
  t.reset();

  size_t numThreads = 1024;

  setBoundsAndCubes<<<iDivUp(numOfCubes, numThreads), numThreads>>>(ptr_cubesLower, levelSizeInMC, mcID(lowestCube),
                                                                    ptr_listOfcubesIDXsforMC, ptr_mcBounds, level, ptr_offsetsCubes, numOfCubes);
  hipPeekAtLastError();


  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 1 run time\n"
            << std::endl;
  float kernel1Time = t.elapsed();
  t.reset();

  auto timeAfterFirstStep = high_resolution_clock::now();

  // 2nd kernel

  // offsets for scalars
  int *ptr_maxNumOfScalars;

  // last entry = size of maxScalarsArray
  hipMalloc((void **)&ptr_maxNumOfScalars, (numberOfMC + 1) * sizeof(int));
  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 2 alloc. \n"
            << std::endl;
  t.reset();

  calcMaxNumOfScalars<<<iDivUp(numberOfMC, numThreads), numThreads>>>(numberOfMC, ptr_mcBounds, ptr_maxNumOfScalars);

  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 2 run time\n"
            << std::endl;

  float kernel2Time = t.elapsed();
  t.reset();

  thrust::device_ptr<int> thr_ptr_maxNumOfScalars = thrust::device_pointer_cast(ptr_maxNumOfScalars);

  thrust::exclusive_scan(thr_ptr_maxNumOfScalars, thr_ptr_maxNumOfScalars + numberOfMC + 1, thr_ptr_maxNumOfScalars);
  ptr_maxNumOfScalars = thrust::raw_pointer_cast(thr_ptr_maxNumOfScalars);

  int totalNumberOfScalars;
  hipMemcpy(&totalNumberOfScalars, ptr_maxNumOfScalars + numberOfMC, sizeof(int), hipMemcpyDeviceToHost);

  std::cout << __LINE__ << " " << t.elapsed() << "s prefixsum run time\n"
            << std::endl;
  float prefixSumTime = t.elapsed();
  t.reset();

  auto timeAfterSecondStep = high_resolution_clock::now();

  // 3rd kernel
  Brick *ptr_mcBricks;
  int *ptr_scalarsArray;
  int *ptr_resultScalarsArray;

  resultScalarArray = new int[totalNumberOfScalars];

  std::vector<Brick> mcBricks(numberOfMC, level);
  mcBricks.reserve(numberOfMC);

  hipMalloc((void **)&ptr_scalarsArray, numOfCubes * 8 * sizeof(int));
  hipMalloc((void **)&ptr_mcBricks, numberOfMC * sizeof(Brick));
  hipMalloc((void **)&ptr_resultScalarsArray, totalNumberOfScalars * sizeof(int));
  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 3 alloc. \n"
            << std::endl;
  t.reset();

  hipMemcpy(ptr_scalarsArray, &scalarsArray[0], 8 * numOfCubes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ptr_mcBricks, &mcBricks[0], numberOfMC * sizeof(Brick), hipMemcpyHostToDevice);
  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 3 copy\n"
            << std::endl;
  t.reset();

  createAndFillBricks<<<iDivUp(numberOfMC, numThreads), numThreads>>>(ptr_cubesLower, ptr_mcBounds, ptr_listOfcubesIDXsforMC, ptr_scalarsArray, 
                                                                  ptr_mcBricks, level, ptr_offsetsCubes, numberOfMC, ptr_maxNumOfScalars, ptr_resultScalarsArray);
  

  std::cout << __LINE__ << " " << t.elapsed() << "s kernel 3 run time\n" << std::endl;
  float kernel3Time= t.elapsed();
  t.reset();

  hipMemcpy(&mcBricks[0], ptr_mcBricks, numberOfMC * sizeof(Brick), hipMemcpyDeviceToHost);
  hipMemcpy(&resultScalarArray[0], ptr_resultScalarsArray, totalNumberOfScalars * sizeof(int), hipMemcpyDeviceToHost);
  std::cout << __LINE__ << " " << t.elapsed() << "s copy result to CPU \n" << std::endl;
  t.reset();

  hipFree(ptr_cubesLower);
  hipFree(ptr_mcBounds);
  hipFree(ptr_listOfcubesIDXsforMC);
  hipFree(ptr_mcBricks);
  hipFree(ptr_scalarsArray);
  hipFree(ptr_offsetsCubes);
  hipFree(ptr_maxNumOfScalars);
  hipFree(ptr_resultScalarsArray);

  // set pointers: Brick-> resultScalarArray
  for (size_t i = 0; i < numberOfMC; i++){
    mcBricks[i].scalarIDs = &resultScalarArray[mcBricks[i].offset];
  }

  auto timeAfterThirdStep = high_resolution_clock::now();

  if (PRINT_EVERY_BRICK_SCALAR){
    int brickNum = 0;
    int brickOffset = mcBricks[0].offset;
    printf("Scalars: ");
    for (int i = 0; i < totalNumberOfScalars; i++){
      if (i == brickOffset){
        // new brick beginns
        std::cout << "\n printing scalars for brick.lower = :" << mcBricks[brickNum].lower << std::endl;
        brickNum++;
        brickOffset = mcBricks[brickNum].offset;
      }
      printf("%d ", resultScalarArray[i]);
    }
    printf("\n---------------------------\n");
  }

  float timeForStep1 = (timeAfterFirstStep - start).count() / 1000000000.0; 
  float timeForStep2 = (timeAfterSecondStep - timeAfterFirstStep).count() / 1000000000.0;
  float timeForStep3 = (timeAfterThirdStep- timeAfterSecondStep).count() / 1000000000.0; 
  float totalStepTime = (timeAfterThirdStep - start).count() / 1000000000.0;

  std::cout << "Time taken by first step: "
            << timeForStep1 << " s" << std::endl;

  std::cout << "Time taken by second step: "
            << timeForStep2 << " s" << std::endl;

  std::cout << "Time taken by third step: "
            << timeForStep3 << " s" << std::endl;

  std::cout << "Time taken by entire function: "
            << totalStepTime << " s" << std::endl;

  if (PRINT_STAT){
    writeStat(level, numberOfMC, numOfCubes, kernel1Time, kernel2Time, kernel3Time, kernel1Time+kernel2Time+kernel3Time+prefixSumTime, 
                prefixSumTime, timeForStep1, timeForStep2, timeForStep3, totalStepTime);
  }

  return mcBricks;
}

void writeQuadOBJ(std::ostream &out,
                  vec3f base,
                  vec3f du,
                  vec3f dv){
  vec3f v00 = base;
  vec3f v01 = base + du;
  vec3f v11 = base + du + dv;
  vec3f v10 = base + dv;
  out << "v " << v00.x << " " << v00.y << " " << v00.z << std::endl;
  out << "v " << v01.x << " " << v01.y << " " << v01.z << std::endl;
  out << "v " << v10.x << " " << v10.y << " " << v10.z << std::endl;
  out << "v " << v11.x << " " << v11.y << " " << v11.z << std::endl;
  out << "f -1 -2 -4 -3" << std::endl;
}

void writeOBJ(std::ostream &out, const box3f &box){
  vec3f dx(box.size().x, 0.f, 0.f);
  vec3f dy(0.f, box.size().y, 0.f);
  vec3f dz(0.f, 0.f, box.size().z);
  writeQuadOBJ(out, box.lower, dx, dy);
  writeQuadOBJ(out, box.lower, dx, dz);
  writeQuadOBJ(out, box.lower, dy, dz);
  writeQuadOBJ(out, box.upper, -dx, -dy);
  writeQuadOBJ(out, box.upper, -dx, -dz);
  writeQuadOBJ(out, box.upper, -dy, -dz);
}

void writeBIN(std::ostream &out, const Brick &brick){
  out.write((const char *)&brick.lower, sizeof(brick.lower));
  out.write((const char *)&brick.level, sizeof(brick.level));
  out.write((const char *)&brick.numCubes, sizeof(brick.numCubes));
  out.write((const char *)brick.scalarIDs, brick.numScalars * sizeof(int));
}

void makeGridsFor(const std::string &fileName){
  std::cout << "==================================================================" << std::endl;
  std::cout << "making grids for " << fileName << std::endl;
  std::cout << "==================================================================" << std::endl;
  const char *ext = strstr(fileName.c_str(), "_");
  if (!ext)
    throw std::runtime_error("'" + fileName + "' is not a cubes file!?");
  while (const char *next = strstr(ext + 1, "_"))
    ext = next;
  int level;
  int rc = sscanf(ext, "_%i.cubes", &level);
  if (rc != 1)
    throw std::runtime_error("'" + fileName + "' is not a cubes file!?");
  
  std::vector<Cube> cubes;
  std::ifstream in(fileName, std::ios::binary);
  
  gridlets::timer t2;
  while (!in.eof())
  {
    Cube cube;
    in.read((char *)&cube, sizeof(cube));
    cubes.push_back(cube);
  }

  std::cout << t2.elapsed() << "s for copying cubes from file" << std::endl;

  // scalars for each brick are stored here consecutively
  int *resultScalarArray = NULL;

  std::vector<Brick> bricks = makeBricksForLevel(level, cubes, resultScalarArray);

#if 1
  int numBricksGenerated = 0;
  int numCubesInBricks = 0;
  int numScalarsInBricks = 0;
  for (auto &brick : bricks){
    if(brick.numCubes.x != 0){
      numBricksGenerated++;
      numCubesInBricks += brick.numCubes.x * brick.numCubes.y * brick.numCubes.z;
      numScalarsInBricks += brick.numScalars;
    }
  }
  PRINT(numBricksGenerated);
  PRINT(numCubesInBricks);
  PRINT(numScalarsInBricks);
  static int totalBricksGenerated = 0;
  static int totalCubesInBricks = 0;
  static int totalScalarsInBricks = 0;

  totalBricksGenerated += numBricksGenerated;
  totalCubesInBricks += numCubesInBricks;
  totalScalarsInBricks += numScalarsInBricks;

  PRINT(totalBricksGenerated);
  PRINT(totalCubesInBricks);
  PRINT(totalScalarsInBricks);
  PRINT(prettyNumber(totalBricksGenerated));
  PRINT(prettyNumber(totalCubesInBricks));
  PRINT(prettyNumber(totalScalarsInBricks));

  static int fileID = 0;
  std::ofstream out;
  
  std::string outName = "./outputGrids/cuda_k3_level_"+std::to_string(level)+".grids";

  if (fileID++ == 0)
    out.open(outName, std::ios_base::binary);
  else
    out.open(outName, std::ios_base::binary | std::ios_base::app);
  for (auto &brick : bricks){
    if(brick.numCubes.x != 0){
      writeBIN(out, brick);
    }
  }
#else
  std::ofstream out("./outputGrids/out.obj");
  for (auto &brick : bricks)
  {
    writeOBJ(out, worldBounds(brick));
  }
#endif

  delete[] resultScalarArray;
}

int main(int ac, char **av){
  gridlets::timer t_sum;

  if(PRINT_STAT){
    std::ofstream outFile;
    outFile.open ("stat.txt", std::ofstream::out | std::ofstream::app);

    if(outFile.is_open()){
      outFile << __FILE__ << std::endl;
    }
    else{
      std::cout << "Error opening file!" << std::endl;
    }

    for (int i = 1; i < ac; i++){
      outFile << av[i] << std::endl; 
      makeGridsFor(av[i]);
    }    
  }
  else{
    for (int i = 1; i < ac; i++){
      makeGridsFor(av[i]);
    }
  }
    
  std::cout << t_sum.elapsed() << "s for all levels" << std::endl; 
  t_sum.reset();
}
